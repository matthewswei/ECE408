#include "hip/hip_runtime.h"
// MP 5.1 Reduction
// Given a list of length n
// Output its sum = lst[0] + lst[1] + ... + lst[n-1];

#include <wb.h>

#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)
  
__global__ void total(float *input, float *output, int len) {
  //@@ Load a segment of the input vector into shared memory
  //@@ Traverse the reduction tree
  //@@ Write the computed sum of the block to the output vector at the
  //@@ correct index

  __shared__ float shared[BLOCK_SIZE];

  int bx = blockIdx.x;
  int tx = threadIdx.x;

  // Loading input vector into shared memory
  int index = bx*BLOCK_SIZE + tx;
  float val = 0;
  if (index<len) {
    val = input[index];
  }
  shared[tx] = val;
  __syncthreads();

  // Computing sum through reduction
  for (int i = 1; i<BLOCK_SIZE; i*=2) {
    if (tx%(i*2)==0) {
      shared[tx]+=shared[tx+i];
    }
    __syncthreads();
  }

  // Writing computed sum of block onto correct output vector index
  if (tx==0) {
    output[bx] = shared[0];
  }

}

int main(int argc, char **argv) {
  int ii;
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  float *deviceInput;
  float *deviceOutput;
  int numInputElements;  // number of elements in the input list
  int numOutputElements; // number of elements in the output list

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput =
      (float *)wbImport(wbArg_getInputFile(args, 0), &numInputElements);

  // numOutputElements = (numInputElements - 1) / (BLOCK_SIZE << 1) + 1;
  numOutputElements = ceil(1.0*numInputElements/BLOCK_SIZE);
  hostOutput = (float *)malloc(numOutputElements * sizeof(float));

  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The number of input elements in the input is ",
        numInputElements);
  wbLog(TRACE, "The number of output elements in the input is ",
        numOutputElements);

  int sizeInput = sizeof(float)*numInputElements;
  int sizeOutput = sizeof(float)*numOutputElements;

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void**)&deviceInput, sizeInput);
  hipMalloc((void**)&deviceOutput, sizeOutput);

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceInput, hostInput, sizeInput, hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input memory to the GPU.");
  //@@ Initialize the grid and block dimensions here
  dim3 dimGrid(numOutputElements, 1, 1);
  dim3 dimBlock(BLOCK_SIZE, 1, 1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  total<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, numInputElements);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, deviceOutput, sizeOutput, hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying output memory to the CPU");

  /***********************************************************************
   * Reduce output vector on the host
   * NOTE: One could also perform the reduction of the output vector
   * recursively and support any size input.
   * For simplicity, we do not require that for this lab!
   ***********************************************************************/
  for (ii = 1; ii < numOutputElements; ii++) {
    hostOutput[0] += hostOutput[ii];
  }

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceOutput);
  hipFree(deviceInput);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, 1);

  free(hostInput);
  free(hostOutput);

  return 0;
}
